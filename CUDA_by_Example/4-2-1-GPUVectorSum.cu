#include "hip/hip_runtime.h"
#include "common/book.h"


#define N 10

__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

int main( void ) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the GPU
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

	// fill the arrays 'a' and 'b' on the CPU
	for (int ii = 0; ii < N; ii++) {
		a[ii] = -ii;
		b[ii] = ii * ii;
	}

	// Copy the arrays 'a' and 'b' to the GPU
	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ) );

	add<<<N, 1>>>( dev_a, dev_b, dev_c );

	// Copy the array 'c' back from the GPU to the CPU
	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

	// Display the results
	for (int ii = 0; ii < N; ii++ ) {
		printf( "%d + %d = %d\n", a[ii], b[ii], c[ii] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;

}
