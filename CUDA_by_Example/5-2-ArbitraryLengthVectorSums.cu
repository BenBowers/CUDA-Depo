#include "hip/hip_runtime.h"
#include "common/book.h"

#define N 10

__global__ void add( int *a, int *b, int *c ) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < N ) {
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main( void )  {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// Allocate the memory on the GPU

	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

	// fill the arrays 'a' and 'b' on the CPU
	for (int ii = 0; ii < N; ii++) {
		a[ii] = ii;
		b[ii] = ii * ii;
	}

	HANDLE_ERROR( hipMemcpy( dev_a,
				a,
				N * sizeof(int),
				hipMemcpyHostToDevice ) );


	HANDLE_ERROR( hipMemcpy( dev_b,
				b,
				N * sizeof(int),
				hipMemcpyHostToDevice ) );
	add<<<1,N>>>( dev_a, dev_b, dev_c );

	HANDLE_ERROR( hipMemcpy( c,
				dev_c,
				N * sizeof(int),
				hipMemcpyDeviceToHost ) );

	// display the results
	for (int ii = 0; ii < N; ii++)
	{
		printf( "%d + %d = %d\n", a[ii], b[ii], c[ii] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}
